#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_knn.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

#include <gunrock/app/knn/knn_app.cu>
#include <gunrock/app/test_base.cuh>

// JSON includes
#include <gunrock/util/info_rapidjson.cuh>

using namespace gunrock;

namespace APP_NAMESPACE = app::knn;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t operator()(util::Parameters& parameters, VertexT v, SizeT s,
                         ValueT val) {
    // CLI parameters
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    // Get number of nearest neighbors, default k = 10
    SizeT k = parameters.Get<int>("k");
    // Get x reference point, default point_id = 0
    VertexT point_x = parameters.Get<VertexT>("x");
    // Get y reference point, default point_id = 0
    VertexT point_y = parameters.Get<VertexT>("y");
    // Get number of neighbors two close points should share
    SizeT eps = parameters.Get<SizeT>("eps");
    // Get the min density
    SizeT min_pts = parameters.Get<SizeT>("min-pts");

    if (min_pts >= k) {
      util::PrintMsg("Min pts should be smaller than k", true);
      return (hipError_t)1;
    }

    util::PrintMsg("Reference point is (" + std::to_string(point_x) + ", " +
                       std::to_string(point_y) + "), k = " + std::to_string(k) +
                       ", eps = " + std::to_string(eps) +
                       +", min-pts = " + std::to_string(min_pts) + "\n",
                   !quiet);

    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    GraphT graph;

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, graph));
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // Reference result on CPU
    SizeT* ref_cluster = NULL;

    SizeT* h_cluster = (SizeT*)malloc(sizeof(SizeT) * graph.nodes);
    SizeT* h_core_point_counter = (SizeT*)malloc(sizeof(SizeT));
    SizeT* h_cluster_counter = (SizeT*)malloc(sizeof(SizeT));

    SizeT* ref_knns = NULL;
    SizeT* h_knns = (SizeT*)malloc(sizeof(SizeT) * graph.nodes * k);

    if (!quick) {
      // Init datastructures for reference result on GPU
      ref_cluster = (SizeT*)malloc(sizeof(SizeT) * graph.nodes);
      for (auto i = 0; i < graph.nodes; ++i) ref_cluster[i] = i;

      ref_knns = (SizeT*)malloc(sizeof(SizeT) * graph.nodes * k);

      // If not in `quick` mode, compute CPU reference implementation
      util::PrintMsg("__________________________", !quiet);
      util::PrintMsg("______ CPU Reference _____", !quiet);

      float elapsed =
          app::knn::CPU_Reference(graph.csr(), k, eps, min_pts, point_x,
                                  point_y, ref_knns, ref_cluster, quiet);

      util::PrintMsg(
          "--------------------------\n Elapsed: " + std::to_string(elapsed),
          !quiet);
      util::PrintMsg("__________________________", !quiet);
      parameters.Set("cpu-elapsed", elapsed);
    }

    std::vector<std::string> switches{"advance-mode"};

    GUARD_CU(app::Switch_Parameters(
        parameters, graph, switches,
        [k, eps, min_pts, h_knns, ref_knns, h_cluster, h_core_point_counter,
         h_cluster_counter,
         ref_cluster](util::Parameters& parameters, GraphT& graph) {
          return app::knn::RunTests(parameters, graph, k, eps, min_pts, h_knns,
                                    ref_knns, h_cluster, ref_cluster,
                                    h_core_point_counter, h_cluster_counter,
                                    util::DEVICE);
        }));

    if (!quick) {
      delete[] ref_cluster;
      delete[] ref_knns;
    }

    delete[] h_cluster;

    return retval;
  }
};

int main(int argc, char** argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test knn");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::knn::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
                           app::SIZET_U32B | app::SIZET_U64B |
                           app::VALUET_S64B | app::DIRECTED | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
